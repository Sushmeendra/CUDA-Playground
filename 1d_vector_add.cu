#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>


const int N = 16384;
const int THREADS_PER_BLOCK = 512;
const int NUM_BLOCKS = (N + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK;

/* Running one thread in each block */
__global__ void add_blocks (int *a, int *b, int *c)
{
  c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

/* Running multiple threads in one block */
__global__ void add_threads (int *a, int *b, int *c)
{
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}


__global__ void add_threads_blocks (int *a, int *b, int *c, int n) {
  
  int index = threadIdx.x * blockIdx.x * threadIdx.x;

  if (index < n) {
    c[index] = a[index] + b[index];
  }
}

int main(void) 
{
  int *a, *b, *c; /* Host (CPU) copies of a, b, c */
  int *d_a, *d_b, *d_c; /* Device (GPU) copies of a, b, c */
  size_t size = N * sizeof(int);


  /* Allocate memory in device */
  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_b, size);
  hipMalloc((void **) &d_c, size);

  /* Allocate memory in host */
  a = (int *) malloc(size);
  b = (int *) malloc(size);
  c = (int *) malloc(size);

  /* Allocate random data in vectors a and b (inside host) */
  for (int i = 0; i < N; ++i) 
  {
    a[i] = rand();
    b[i] = rand();
  }

  /* Copy data to device (GPU) */
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  /* Launching add() kernel on device with N blocks and 1 thread */
  add_blocks<<<N,1>>>(d_a, d_b, d_c);
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  /* Sanity Check */
  for (int i = 0; i < N; ++i) {
    assert(c[i] == a[i] + b[i]);
  }
  printf("Version with %d blocks executed succesfully!\n", N);

  /* Launching add() kernel on device with 1 block and N threads */
  add_threads<<<1,N>>>(d_a, d_b, d_c);
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  /* Sanity Check */
  for (int i = 0; i < N; ++i) {
    assert(c[i] == a[i] + b[i]);
  }
  printf("Version with %d threads executed succesfully!\n", N);

  /* Launching add() kernel on device with N threads and NUM_BLOCKS blocks */
  add_threads_blocks<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  /* Sanity Check */
  for (int i = 0; i < N; ++i) {
    assert(c[i] == a[i] + b[i]);
  }
  printf("Version with %d threads/blocks executed succesfully!\n", N);

  /* Clean-up */
  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  return 0;
}
